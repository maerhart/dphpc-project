#include "hip/hip_runtime.h"
// Main function: entry point into GPU main()
// Only this file should use standard MPI implementation like OpenMPI or MPICH,
// functions on the GPU should use gpu_mpi instead.

#include <hip/hip_runtime.h>
#include <mpi.h>

#include <stdio.h>

#include <string>
#include <vector>
#include <iostream>
#include <set>

#include <cxxopts.hpp>

#include "common.h"

#include "cuda_mpi.cuh"

#include "libc_processor.cuh"
#include "global_vars.cuh"

void* copyArgsToUnifiedMemory(int argc, char** argv) {
    // argv is a set of "pointers" to "strings"
    int stringsSize = 0;
    for (int i = 0; i < argc; i++) {
        stringsSize += (strlen(argv[i]) + 1);
    }
    char* argvInUnifiedMemory = NULL;
    int pointersSize = (argc + 1) * sizeof(char*);
    CUDA_CHECK(hipMallocManaged(&argvInUnifiedMemory, pointersSize + stringsSize));
    char** pointers = (char**) argvInUnifiedMemory;
    char* strings = (char*)&pointers[argc+1];
    char* current_string = strings;
    for (int i = 0; i < argc; i++) {
        pointers[i] = current_string;
        strcpy(current_string, argv[i]);
        current_string += (strlen(argv[i]) + 1);
    }
    pointers[argc] = NULL;
    return argvInUnifiedMemory;
}

/*
 * Parse args related to gpumpi: everything after "---gpumpi"
 * Return new argc: everything after "---gpumpi"
 */
int parseGPUMPIArgs(int argc, char** argv, 
    unsigned& blocksPerGrid, unsigned& threadsPerBlock, unsigned& stackSize, unsigned& heapSize) 
{
    int trippleDashPosition = -1;
    for (int i = 0; i < argc; i++) {
        if (strcmp(argv[i], "---gpumpi") == 0) {
            trippleDashPosition = i;
        }
    }
    if (trippleDashPosition == -1) {
        // no tripple dash, say about it and exit
        std::cerr << "You should specify gpumpi related options after '---gpumpi'" << std::endl;

        std::exit(1);
    }

    char** gpumpi_argv = argv + trippleDashPosition;
    int gpumpi_argc = argc - trippleDashPosition;

    cxxopts::Options options("GPU MPI", "GPU MPI");

    options.add_options()
        ("g,blocksPerGrid", "Blocks per grid", cxxopts::value<unsigned>()->default_value("1"))
        ("b,threadsPerBlock", "Threads per block", cxxopts::value<unsigned>()->default_value("1"))
        ("s,stackSize", "Override stack size limit on GPU (bytes)", cxxopts::value<unsigned>()->default_value("1024"))
        ("p,heapSize", "Override heap size limit on GPU (bytes)", cxxopts::value<unsigned>()->default_value("8388608"))
        ("h,help", "Print help text")
        ;


    auto result = options.parse(gpumpi_argc, gpumpi_argv);

    if (result.count("help"))
    {
        std::cout << options.help() << std::endl;
        exit(0);
    }

    blocksPerGrid = result["blocksPerGrid"].as<unsigned>();
    threadsPerBlock = result["threadsPerBlock"].as<unsigned>();
    stackSize = result["stackSize"].as<unsigned>();
    heapSize = result["heapSize"].as<unsigned>();

    return trippleDashPosition;
}

extern __device__ int __gpu_main(int argc, char* argv[]);
extern __device__ void __gpu_init_global_ptrs();

__device__ void** __gpu_global_vars = nullptr;
__device__ void initializeGlobalVars() {
    int j = CudaMPI::sharedState().gridRank();
    for (int i = 0; i < __gpu_num_globals; i++) {
        if (__gpu_global_ptrs[i]) {
            __gpu_global_vars[i + j * __gpu_num_globals] = (void*) malloc(__gpu_global_size[i]);
            memcpy(__gpu_global_vars[i + j * __gpu_num_globals], __gpu_global_ptrs[i], __gpu_global_size[i]);
        } else {
            // static scoped global variables initialized at the point of definition
            __gpu_global_vars[i + j * __gpu_num_globals] = nullptr;
        }
    }
    // I never free this memory, but it is not required since global variables should live for entire program live
}

__global__ void __gpu_main_caller(int argc, char* argv[],
                                    CudaMPI::SharedState* sharedState,
                                    CudaMPI::ThreadPrivateState::Context threadPrivateStateContext)
{
    CudaMPI::setSharedState(sharedState);
    CudaMPI::ThreadPrivateState::Holder threadPrivateStateHolder(threadPrivateStateContext);

    // initialize global variables
    if (CudaMPI::sharedState().gridRank() == 0) {
        __gpu_init_global_ptrs();
        __gpu_global_vars = (void**) malloc(CudaMPI::sharedState().gridSize() * __gpu_num_globals * sizeof(void*));
    }
    CudaMPI::sharedState().gridBarrier();
    initializeGlobalVars();

    int returnValue = __gpu_main(argc, argv);
    if (returnValue != 0) {
        sharedState->returnValue = 1;
    }
}

int main(int argc, char* argv[]) {

    MPI_CHECK(MPI_Init(&argc, &argv));

    int deviceCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    unsigned blocksPerGrid = 0;
    unsigned threadsPerBlock = 0;
    unsigned stackSize = 0;
    unsigned heapSize = 0;

    int argcWithoutGPUMPI = parseGPUMPIArgs(argc, argv, blocksPerGrid, threadsPerBlock, stackSize, heapSize);

    // increase stack size
    CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, stackSize));
    // increase heap size
    CUDA_CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize));


    // convert the argv array into memory inside the an UM allocated buffer
    void* argvInUnifiedMemory = copyArgsToUnifiedMemory(argcWithoutGPUMPI,argv);

    // allocate memory for communication
    CudaMPI::SharedState::Context sharedStateContext;
    sharedStateContext.numThreads = blocksPerGrid * threadsPerBlock;
    sharedStateContext.recvListSize = 16;
    sharedStateContext.numFragments = 256;
    sharedStateContext.fragmentSize = 1024;
    sharedStateContext.numIncomingFragments = 64;

    CudaMPI::SharedState::Holder sharedStateHolder(sharedStateContext);
    CudaMPI::SharedState* sharedState = sharedStateHolder.get();

    //create cuda streams for each device
    std::vector<hipStream_t> cudaStreams(deviceCount);
    for(int i = 0; i < deviceCount; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipStreamCreate(&cudaStreams[i]));
    }

    std::vector<CudaMPI::ThreadPrivateState::Context> threadPrivateStateContext(deviceCount);
    for(int device = 0; device < deviceCount; device++) {
        threadPrivateStateContext[device].pendingBufferSize = 20;

        int peakClockKHz;
        CUDA_CHECK(hipDeviceGetAttribute(&peakClockKHz, hipDeviceAttributeClockRate, device));
        threadPrivateStateContext[device].peakClockKHz = peakClockKHz;
    }

    // args passed into kernel function
    std::vector<std::vector<void*>> params(deviceCount);
    for(int i = 0; i < deviceCount; i++) {
        params[i] = {
            (void*)&argcWithoutGPUMPI,
            (void*)&argvInUnifiedMemory,
            (void*)&sharedState,
            (void*)&threadPrivateStateContext[i],
        };
    }

    std::vector<hipLaunchParams> launchParamsList(deviceCount);
    for(int i = 0; i < deviceCount; i++) {
        launchParamsList[i].func = (void*) __gpu_main_caller;
        launchParamsList[i].gridDim = blocksPerGrid;
        launchParamsList[i].blockDim = threadsPerBlock;
        launchParamsList[i].args = params[i].data();
        launchParamsList[i].sharedMem = 0;
        launchParamsList[i].stream = cudaStreams[i];
    }

    hipEvent_t kernelFinishEvent;
    CUDA_CHECK(hipEventCreate(&kernelFinishEvent));

    std::cerr << "GPUMPI: Starting kernel!" << std::endl;
    // here we actually call __gpu_main
    //CUDA_CHECK(hipLaunchCooperativeKernelMultiDevice(launchParamsList.data(), deviceCount));
    for(int i = 0; i < deviceCount; i++) {
        CUDA_CHECK(hipLaunchKernel((void*)__gpu_main_caller, blocksPerGrid, threadsPerBlock, params[i].data(), 0, cudaStreams[i]));
    }
    CUDA_CHECK(hipEventRecord(kernelFinishEvent));

    std::cerr << "GPUMPI: Processing messages from device threads" << std::endl;

    while (hipEventQuery(kernelFinishEvent) == hipErrorNotReady) {
        sharedState->deviceToHostCommunicator.processIncomingMessages([&](void* ptr, size_t size, int threadRank) {
            if (ptr == 0 && size == 0) {
                // nothing to do, this is notification that thread finished execution
            } else {
                process_gpu_libc(ptr, size);
            } 
        });
    }
    std::cerr << "GPUMPI: Kernel finished, stop processing messages from device threads" << std::endl;

    // make sure that everything is ok after kernel launch
    CUDA_CHECK(hipEventQuery(kernelFinishEvent));

    std::cerr << "GPUMPI: Releasing resources" << std::endl;

    // release all resources

    CUDA_CHECK(hipFree(argvInUnifiedMemory));

    MPI_CHECK(MPI_Finalize());

    std::cerr << "GPUMPI: MPI finished!" << std::endl;

    return sharedState->returnValue;
}

