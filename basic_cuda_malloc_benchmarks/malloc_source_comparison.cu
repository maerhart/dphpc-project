#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void sum_values_in_allocated_array(int* array, int range, int* res) {
    int sum = 0;
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = start; i < start + range; i++) {
        sum += (i+array[i]);
    }
    res[start] = sum;
}

__global__ void sum_values_and_allocate(int size, int* res) {
    int sum = 0;
    int *array;
    
    hipMalloc(&array, size);
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i < size; i++) {
        sum += (i+array[i]);
    }
    hipFree(array);
    res[id] = sum;
}

int main(int argc, char **argv) {
    hipError_t cuda_status;
    printf("%s Starting...\n", argv[0]);
    int coalesced = (atoi(argv[2]) == 1) ? 1 : 0;
    int bytes = 1 << atoi(argv[1]);
    if(coalesced > 0) {
        printf("Benchmarking Coalesced for %d bytes\n", bytes);
    } else {
        printf("Benchmarking non coalesced for %d bytes\n", bytes);
    }
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    int threads_per_block = 1024;
    int blocks = 1024;
    int total_threads = threads_per_block * blocks;

    int* res;
    hipMalloc((void**)&res, total_threads * sizeof(int));
    int allocation_size = bytes;
    int allocation_per_thread = allocation_size / total_threads;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipDeviceSynchronize();
    float ms = 0;
    if(coalesced) {
        hipEventRecord(start);
        sum_values_and_allocate<<<blocks, threads_per_block>>>(allocation_per_thread, res);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        cuda_status = hipDeviceSynchronize();
        printf("cudaMalloc(%d) over %d threads: Time elapsed %f ms\n", allocation_per_thread, total_threads, ms);
    } else {
        hipEventCreate(&start);
        int* array;
        hipMalloc((void**)&array, allocation_size);
        sum_values_in_allocated_array<<<blocks, threads_per_block>>>(array, allocation_per_thread / sizeof(int), res);
        hipFree(array);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        cuda_status = hipDeviceSynchronize();
        printf("cudaMalloc(%d) coalesced: Time elapsed %f ms\n", allocation_size, ms);
    }

    if (cuda_status != hipSuccess) {
        printf("Error: %d\n", cuda_status);
        exit(1);
    }
    hipFree(res);
    return 0;
}
