#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <>
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

//__global__ void simulate_dynamic_coalescing(int* atomic_counters, void* pointers_dynamically_allocated, int num_threads, int num_semaphores, int* global_atomic_counter) {
//    long index = blockIdx.x * blockDim.x + threadIdx.x;
//    int arriveNumber = atomicAdd(global_atomic_counter, 1);
//    int threads_per_semaphore = num_threads / num_semaphores;
//    int array_index = arriveNumber / threads_per_semaphore;
//    bool allocator = (num_threads % threads_per_semaphore) == 0;
//
//    if(allocator) {
//
//    } else {
//	do {
//
//	} while(!pointers_dynamiucally_allocated[array_index])
//    }
//}

__global__ void sum_values_and_allocate(long value, int* res) {
    int val = 0;

    int *array = (int *)malloc(value * sizeof(int));
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(array == NULL) {
        val = 1;
    }
    free(array);
    res[id] = val;
}

__global__ void simulate_dynamic_coalescing(int* atomic_counters_per_block, int** pointers_dynamically_allocated_per_block, long ints_per_block, int* res) {
    int val = 0;
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    long id = blockIdx.x;

    auto block = cooperative_groups::this_thread_block();


    int arriveNumber = atomicAdd(&atomic_counters_per_block[id], 1);
    if(arriveNumber == 0) {
	pointers_dynamically_allocated_per_block[id] = (int *)malloc(ints_per_block * sizeof(int));
	val = (!pointers_dynamically_allocated_per_block[id]) ? 1:0;
    }

    block.sync();

    int leaveNumber = atomicAdd(&atomic_counters_per_block[id], -1);
    if(leaveNumber == 0 && pointers_dynamically_allocated_per_block[id]) {
	free(pointers_dynamically_allocated_per_block[id]);
    }
    res[index] = val;
}

int main(int argc, char **argv) {
    hipError_t cuda_status;
    int coalesced = (atoi(argv[2]) == 1) ? 1 : 0;
    char *simple_output = argv[5];
    if(!simple_output) {
	    printf("%s Starting...\n", argv[0]);
    }
    long ints = 1L << atoi(argv[1]);
    if(!simple_output) {
	if(coalesced > 0) {
	    printf("Benchmarking Coalesced for %ld ints\n", ints);
	} else {
	    printf("Benchmarking non coalesced for %ld ints\n", ints);
	}
    }

    int threads_per_block = 1 << atoi(argv[3]);
    // max value is 1024
    threads_per_block = (threads_per_block > 1024) ? 1024 : threads_per_block;
    int blocks = 1 << atoi(argv[4]);
    // max value of concurrently executable blocks is 192 (largest power of 2 is 128)
    blocks = (blocks > 65535) ? 65535 : blocks;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if(!simple_output) {
        printf("Using Device %d: %s\n", dev, deviceProp.name);
    }
    hipSetDevice(dev);

    int total_threads = threads_per_block * blocks;

    int* res;
    hipMalloc((void**)&res, total_threads * sizeof(int));
    hipMemset(res, 0, total_threads * sizeof(int));
    int* resCPU = (int *) malloc(total_threads * sizeof(int));
    long allocation_size = ints;
    long allocation_per_thread = allocation_size / total_threads;
    long allocation_per_block = allocation_size / blocks;
    allocation_per_thread = (allocation_per_thread < 1) ? 1 : allocation_per_thread;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipDeviceSynchronize();
    float ms = 0;
    if(!coalesced) {
        hipEventRecord(start);
        sum_values_and_allocate<<<blocks, threads_per_block>>>(allocation_per_thread, res);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        cuda_status = hipDeviceSynchronize();
        if(!simple_output) {
            printf("hipMalloc(%ld ints/%ld bytes) over %d threads (%d blocks, %d threads per block): Time elapsed %f ms\n", allocation_per_thread, allocation_per_thread * sizeof(int), total_threads, blocks, threads_per_block, ms);
	} else {
	    printf("%d %d %f ", blocks, threads_per_block, ms);
	}
    } else {
	int** block_array;
	hipMalloc((void***)&block_array, blocks * sizeof(int *));
	int* array;
	hipMalloc((void**)&array, blocks * sizeof(int));
	hipMemset(array, 0, blocks * sizeof(int));
        hipEventRecord(start);
        simulate_dynamic_coalescing<<<blocks, threads_per_block>>>(array, block_array, allocation_per_block, res);
        hipFree(array);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        cuda_status = hipDeviceSynchronize();
        if(!simple_output) {
            printf("hipMalloc(%ld ints/%ld bytes) coalesced (%d blocks, %d threads per block): Time elapsed %f ms\n", allocation_size, allocation_size * sizeof(int), blocks, threads_per_block, ms);
	} else {
	    printf("%d %d %f ", blocks, threads_per_block, ms);
	}
    }
    hipMemcpy(resCPU, res, total_threads * sizeof(int), hipMemcpyDeviceToHost);
    int malloc_failures = 0;
    for(int i = 0; i < total_threads; i++) malloc_failures += resCPU[i];

    if(!simple_output) {
        printf("%d threads failed to allocate memory\n", malloc_failures);
    } else {
        printf("%d\n", malloc_failures);
    }

    if (cuda_status != hipSuccess) {
        printf("Error: %d %s\n", cuda_status, hipGetErrorString(cuda_status));
        exit(1);
    }
    hipFree(res);
    return 0;
}
