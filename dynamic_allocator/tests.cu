#include "hip/hip_runtime.h"
#include <iostream>
#include "dynamic_allocator.cu"
#include "warp_malloc.cu"

#define MALLOC malloc_v4
#define FREE free_v4

// allocate one int per thread and set to threadId
__global__ void test(int *resulting_ids) {
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    int* val = (int*) MALLOC(sizeof(int));
    *val = id;
    resulting_ids[id] = *val;
    FREE(val);
}

__global__ void test_different_size(int *resulting_ids) {
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    size_t size = sizeof(int) * (1 + (id % 32));
    int* val = (int*) MALLOC(size);
    val[id % 32] = id;  // write to very end of sement
    resulting_ids[id] = val[id % 32];
    FREE(val);
}

__global__ void test_different_types(int *resulting_ids) {
    // good for alignemnt requirements
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    int choice = id % 5;
    if (choice == 0 && ((char) id) == id) {
        char* val = (char*) MALLOC(sizeof(char));
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else if (choice == 1 && ((short int) id) == id) {
        short int* val = (short int*) MALLOC(sizeof(short int));
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else if (choice == 2) {
        long int* val = (long int*) MALLOC(sizeof(long int)); // size 64 bits
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else if (choice == 300) { // TODO doesn't pass yet
    	// check for 128 bits
        int* val = (int*) MALLOC(128);
    	assert(((long) val) % 128  == 0);
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else if (choice == 400) { // TODO doesn't pass yet
        // check that alignment correct for max_align_t
        int max_size = sizeof(max_align_t);
        int* val = (int*) MALLOC(max_size);
    	assert(((long) val) % max_size == 0);
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else {
        int* val = (int*) MALLOC(sizeof(int));
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    }
}

/**
 * Passes pointers around and doesn't free all at same time
 */
__global__ void test_pass_ptrs(int *resulting_ids) {

    __shared__ int** ptrs;
    // let thread 0 in block allocate array for entire block
    if (threadIdx.x == 0) {
        ptrs = (int**) MALLOC(sizeof(int*) * blockDim.x);
    } 
    __syncthreads();

    // malloc int and share pointer
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    int* val = (int*) MALLOC(sizeof(int));
    ptrs[threadIdx.x] = val;


    // write id if even thread 
    if (id % 2 == 0) {
        *val = id;
        // shuffle pointers around warps

        // map lane i warp j  -> warp i lane j
        // if i >= #warps or j >= #lanes => leave

        int lane = threadIdx.x % 32;
        int warp = threadIdx.x / 32;
        if (lane < blockDim.x / 32 && warp < 32) {
            int temp = lane;
            lane = warp;
            warp = temp;
        }
        
        val = ptrs[warp * 32 + lane];
        resulting_ids[id] = *val;
    }
    __syncthreads();
    if (id % 2 == 0) {
        FREE(val); // free only here in order not to run into conflict with *val = id;
    }

    // do the same for odd threads
    if (id % 2 == 1) {
        *val = id;
        int lane = threadIdx.x % 32;
        int warp = threadIdx.x / 32;
        if (lane < blockDim.x / 32 && warp < 32) {
            int temp = lane;
            lane = warp;
            warp = temp;
        }
        val = ptrs[warp * 32 + lane];
        resulting_ids[id] = *val;
    }
    __syncthreads();
    if (id % 2 == 1) {
        FREE(val);
    }
}


void run_test(const std::string& name, int blocks, int threads_per_block, void(*kernel)(int*)) {
    std::cout << "Running " << name << " ...  ";

    int total_threads = blocks * threads_per_block;

    int resulting_ids[total_threads];
    int *d_resulting_ids;
    hipMalloc(&d_resulting_ids, total_threads*sizeof(int));

    kernel<<<blocks, threads_per_block>>>(d_resulting_ids);
    hipDeviceSynchronize(); // to allow for printf in kernel code

    // check for error
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
       std::cout << "CUDA ERROR: " << hipGetErrorString(err) << std::endl;
       exit(-1);
    }

    hipMemcpy(resulting_ids, d_resulting_ids, total_threads*sizeof(int), hipMemcpyDeviceToHost);

    // sum up all ids, should match to the sum from 0 to total_threads
    bool passed = true;
    int sum = 0;
    for (int i = 0; i < total_threads; ++i) {
        sum += resulting_ids[i];
    }
    // sum up 0 to total_threads
    passed = sum == (total_threads - 1) * (total_threads) / 2;

    if (passed) {
        std::cout << "Tests passed" << std::endl;
    }
    else {
        std::cout << "Failed" << std::endl;
    }
    
}


int main(int argc, char* argv[]) {
    // run some simple unit tests, only in debug mode!
    int blocks = 100;
    int threads_per_block = 32;
    run_test("basic          ", blocks, threads_per_block, test);
    run_test("different sizes", blocks, threads_per_block, test_different_size);
    run_test("different types", blocks, threads_per_block, test_different_types);
    run_test("pass ptrs      ", blocks, threads_per_block, test_pass_ptrs);


    return 0;
}
