#include "hip/hip_runtime.h"
#include <iostream>
#include "dynamic_allocator.cu"

// allocate one int per thread and set to threadId
__global__ void test(int *resulting_ids) {

    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    int* val = (int*)malloc_v2(sizeof(int));
    *val = id;
    resulting_ids[id] = *val;
	free_v2(val);
}

int main(int argc, char* argv[]) {
    // run some simple unit tests, only in debug mode!
    int blocks = 100;
    int threads_per_block = 32;
    int total_threads = blocks * threads_per_block;

    int resulting_ids[total_threads];
    int *d_resulting_ids;
    hipMalloc(&d_resulting_ids, total_threads*sizeof(int));

    test<<<blocks, threads_per_block>>>(d_resulting_ids);
    hipDeviceSynchronize(); // to allow for printf in kernel code
    hipMemcpy(resulting_ids, d_resulting_ids, total_threads*sizeof(int), hipMemcpyDeviceToHost);

    // sum up all ids, should match to the sum from 0 to total_threads
    bool passed = true;
    int sum = 0;
    for (int i = 0; i < total_threads; ++i) {
        sum += resulting_ids[i];
    }
    // sum up 0 to total_threads
    passed = sum == (total_threads - 1) * (total_threads) / 2;

    if (passed) {
        std::cout << "Tests passed" << std::endl;
    }
    else {
        std::cout << "Failed" << std::endl;
    }

    return 0;
}
