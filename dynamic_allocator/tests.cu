#include "hip/hip_runtime.h"
#include <iostream>
#include "../gpu_libs/gpu_malloc/dynamic_allocator.cu"
#include "../gpu_libs/gpu_malloc/warp_malloc.cu"
#include "../gpu_libs/gpu_malloc/dyn_malloc.cu"

#define COALESCE true

#define MALLOC dyn_malloc
#define FREE dyn_free

// allocate one int per thread and set to threadId
__global__ void test(int *resulting_ids) {
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    int* val = (int*) MALLOC(sizeof(int), COALESCE);
    *val = id;
    resulting_ids[id] = *val;
    FREE(val);
}

__global__ void test_floats(int *resulting_ids) {
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    size_t size = sizeof(float) * 10;
    float* val = (float*) MALLOC(size, COALESCE);
    val[0] = id;  // write to start of segment
    resulting_ids[id] = val[0];
    FREE(val);
}

__global__ void test_different_size(int *resulting_ids) {
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    size_t size = sizeof(int) * (1 + (id % 32));
    int* val = (int*) MALLOC(size, COALESCE);
    val[id % 32] = id;  // write to very end of segment
    resulting_ids[id] = val[id % 32];
    FREE(val);
}

__global__ void test_different_types(int *resulting_ids) {
    // good for alignemnt requirements
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    int choice = id % 5;
    if (choice == 0 && ((char) id) == id) {
        char* val = (char*) MALLOC(sizeof(char), COALESCE);
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else if (choice == 1 && ((short int) id) == id) {
        short int* val = (short int*) MALLOC(sizeof(short int), COALESCE);
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else if (choice == 2) {
        long int* val = (long int*) MALLOC(sizeof(long int), COALESCE); // size 64 bits
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else if (choice == 3) { // TODO do we need this?
    	// check for 128 bits
        int* val = (int*) MALLOC(16, COALESCE);
    	assert(((long) val) % 16 == 0);
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else if (choice == 456) { // TODO normal malloc wouldn't even pass this. why?
        // check that alignment correct for max_align_t
        int max_size = sizeof(max_align_t);
        int* val = (int*) MALLOC(max_size, COALESCE);
    	assert(((long) val) % max_size == 0);
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    } else {
        int* val = (int*) MALLOC(sizeof(int), COALESCE);
        *val = id;
        resulting_ids[id] = *val;
        FREE(val);
    }
}

/**
 * Passes pointers around and doesn't free all at same time
 */
__global__ void test_pass_ptrs(int *resulting_ids) {

    __shared__ int** ptrs;
    // let thread 0 in block allocate array for entire block
    if (threadIdx.x == 0) {
        ptrs = (int**) MALLOC(sizeof(int*) * blockDim.x, COALESCE);
    } 
    __syncthreads();

    // malloc int and share pointer
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    int* val = (int*) MALLOC(sizeof(int), COALESCE);
    assert(val != NULL);
    ptrs[threadIdx.x] = val;
    __syncthreads();

    // NOTE lane and warp get modified for the shuffling of values
    int lane = threadIdx.x % 32;
    int warp = threadIdx.x / 32;
    bool even_thread = lane % 2 == 0 && warp % 2 == 0;

    // write id if everything even
    if (even_thread) {
        *val = id;
    }
    __syncthreads(); // make sure that pointer written before read
    if (even_thread) {
        // shuffle pointers around warps

        // map lane i warp j  -> warp i lane j
        // if i >= #warps or j >= #lanes => leave

        if (lane < blockDim.x / 32 && warp < 32) {
            int temp = lane;
            lane = warp;
            warp = temp;
        }
        
        val = ptrs[warp * 32 + lane];
        resulting_ids[id] = *val;
        FREE(val);
    }

    // do the same for odd threads
    if (!even_thread) {
        *val = id;
    }
    __syncthreads();
    if (!even_thread) {
        if (lane < blockDim.x / 32 && warp < 32) {
            int temp = lane;
            lane = warp;
            warp = temp;
        }
        val = ptrs[warp * 32 + lane];
        resulting_ids[id] = *val;
        FREE(val);
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        FREE(ptrs);
    }
}


void run_test(const std::string& name, int blocks, int threads_per_block, void(*kernel)(int*)) {
    std::cout << "Running " << name << " ...  ";

    int total_threads = blocks * threads_per_block;

    int resulting_ids[total_threads];
    int *d_resulting_ids;
    hipMalloc(&d_resulting_ids, total_threads*sizeof(int));

    kernel<<<blocks, threads_per_block>>>(d_resulting_ids);
    hipDeviceSynchronize(); // to allow for printf in kernel code

    // check for error
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
       std::cout << "CUDA ERROR: " << hipGetErrorString(err) << std::endl;
       exit(-1);
    }

    hipMemcpy(resulting_ids, d_resulting_ids, total_threads*sizeof(int), hipMemcpyDeviceToHost);

    // sum up all ids, should match to the sum from 0 to total_threads
    bool passed = true;
    int sum = 0;
    for (int i = 0; i < total_threads; ++i) {
        sum += resulting_ids[i];
    }
    // sum up 0 to total_threads
    passed = sum == (total_threads - 1) * (total_threads) / 2;

    if (passed) {
        std::cout << "Tests passed" << std::endl;
    }
    else {
        std::cout << "Failed" << std::endl;
    }
    
}


int main(int argc, char* argv[]) {
    // run some simple unit tests, only in debug mode!
    int blocks = 12;
    int threads_per_block = 1024;
    run_test("basic          ", blocks, threads_per_block, test);
    run_test("different sizes", blocks, threads_per_block, test_different_size);
    run_test("different types", blocks, threads_per_block, test_different_types);
    run_test("pass ptrs      ", blocks, threads_per_block, test_pass_ptrs);

    return 0;
}
