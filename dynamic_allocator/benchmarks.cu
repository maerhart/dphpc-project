#include "hip/hip_runtime.h"
#include <iostream>
#include "dynamic_allocator.cu"
#include "../strided_benchmarks/run_benchmark.cu"


__global__ void strided_write_baseline(int num_floats, clock_t* runtime) {
	int id = (blockIdx.x*blockDim.x + threadIdx.x);
	float* ptr = (float*)malloc_baseline(num_floats * sizeof(float));
	//printf("%ld\n", ptr);
	__syncthreads();
	clock_t start_time = clock();

	for (int i = 0; i < num_floats; i++) {
		ptr[i] = id + i;
	}

	clock_t end_time = clock();
	__syncthreads();
	//printf("%ld\n", end_time - start_time);
	//runtime[id] = (double)(end_time - start_time) / (double)CLOCKS_PER_SEC;
	runtime[id] = end_time - start_time;
	free_baseline(ptr);
}

__global__ void strided_write_v1(int num_floats, clock_t* runtime) {
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
	float* ptr = (float*)malloc_v1(num_floats * sizeof(float));
    // check if pointers overlap with runtime
	//printf("Runtime ptr: %p, Malloc ptr: %p\n", runtime, ptr);

	//printf("%p\n", (void*)ptr);
	__syncthreads();
	clock_t start_time = clock();
	
    for (int i = 0; i < num_floats; i++) {
        ptr[i] = id + i;
    }

    clock_t end_time = clock();
	__syncthreads();
    //printf("%ld\n", end_time - start_time);
	//runtime[id] = (double)(end_time - start_time) / (double)CLOCKS_PER_SEC;
	runtime[id] = end_time - start_time;
	//printf("%f\n", runtime[id]);
    free_v1(ptr);
}

__global__ void strided_write_v2(int num_floats, clock_t* runtime) {
    int id = (blockIdx.x*blockDim.x + threadIdx.x);
    float* ptr = (float*)malloc_v2(num_floats * sizeof(float));
    // check if pointers overlap with runtime
    //printf("Runtime ptr: %p, Malloc ptr: %p\n", runtime, ptr);

    //printf("%p\n", (void*)ptr);
    __syncthreads();
    clock_t start_time = clock();

    for (int i = 0; i < num_floats; i++) {
        ptr[i] = id + i;
    }

    clock_t end_time = clock();
    __syncthreads();
    //printf("%ld\n", end_time - start_time);
    //runtime[id] = (double)(end_time - start_time) / (double)CLOCKS_PER_SEC;
    runtime[id] = end_time - start_time;
    //printf("%f\n", runtime[id]);
    free_v2(ptr);
}

void print_arr(double* arr, int len) {
	for (int i = 0; i < len; i++) {
		std::cout << arr[i] << " ";
	}
	std::cout << std::endl;
}

int main(int argc, char **argv) {
	
	// read args
	int blocks = atoi(argv[1]);
	int threads_per_block = atoi(argv[2]);
	int num_runs = atoi(argv[3]);
	int num_warmup = atoi(argv[4]);
	int num_floats = atoi(argv[5]);

	// setup measurement arrays
	double mean_runtimes[num_runs];
	double max_runtimes[num_runs];
	

	/*
	// run benchmarks
	double mean = 0, max = 0;
	int total_threads = blocks * threads_per_block;
	
	for (int i = 0; i < num_runs; ++i) {
		double* runtime_per_thread;
		//double* d_runtime_per_thread;
		int size_runtimes = total_threads * sizeof(double);
		//runtime_per_thread = (double*)malloc(size_runtimes);
		hipMallocManaged(&runtime_per_thread, size_runtimes);

		hipDeviceSetLimit(hipLimitMallocHeapSize, 1000000000); // 1GB
		strided_write_baseline<<<blocks, threads_per_block>>>(num_floats, runtime_per_thread);
		hipDeviceSynchronize();

		//hipMemcpy(runtime_per_thread, d_runtime_per_thread, size_runtimes, hipMemcpyDeviceToHost);
		//hipDeviceSynchronize();
		double mean_run = 0;
		for (int j = 0; j < total_threads; ++j) {
			mean_run += runtime_per_thread[j];
			//printf("%f\n", runtime_per_thread[j]);
		}
		mean_run /= total_threads;

		mean += mean_run;
	}
	mean /= num_runs;
	std::cout << mean << std::endl;
	*/	


	hipDeviceSetLimit(hipLimitMallocHeapSize, 1000000000); // 1GB
	
	// run baseline
	run_benchmark(num_runs, num_warmup, mean_runtimes, max_runtimes, blocks, threads_per_block,
			[num_floats](clock_t* runtimes, int b, int t) -> void {
				strided_write_baseline<<<b, t>>>(num_floats, runtimes);
			}
		     );
	//print_arr(mean_runtimes, num_runs);

	double mean_baseline = 0;
	for (int i = 0; i < num_runs; ++i) {
		mean_baseline += mean_runtimes[i];
	}
	mean_baseline /= num_runs;
	std::cout << mean_baseline << std::endl;
	


	// run v1
    run_benchmark(num_runs, num_warmup, mean_runtimes, max_runtimes, blocks, threads_per_block,
            [num_floats](clock_t* runtimes, int b, int t) -> void {
                strided_write_v1<<<b, t>>>(num_floats, runtimes);
            }
             );
	hipDeviceSynchronize();
    //print_arr(mean_runtimes, num_runs);	

	double mean_v1 = 0;
    for (int i = 0; i < num_runs; ++i) {
        mean_v1 += mean_runtimes[i];
    }
    mean_v1 /= num_runs;
    std::cout << mean_v1 << std::endl;


	// run v2
    run_benchmark(num_runs, num_warmup, mean_runtimes, max_runtimes, blocks, threads_per_block,
            [num_floats](clock_t* runtimes, int b, int t) -> void {
                strided_write_v2<<<b, t>>>(num_floats, runtimes);
            }
             );
    hipDeviceSynchronize();
    //print_arr(mean_runtimes, num_runs);

    double mean_v2 = 0;
    for (int i = 0; i < num_runs; ++i) {
        mean_v2 += mean_runtimes[i];
    }
    mean_v2 /= num_runs;
    std::cout << mean_v2 << std::endl;

	
	return 0;
}
