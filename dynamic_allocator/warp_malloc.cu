
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <utility>


__device__ const int WARP_SIZE = 32;
__device__ const size_t MAX_HEADER_PAD = sizeof(max_align_t) - 1;

__forceinline__ __device__ unsigned lane_id_asm()
{
    unsigned ret;
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}

__forceinline__ __device__ unsigned lane_id() // TODO lane_id not found
{
    unsigned id = threadIdx.x % WARP_SIZE;
    assert(id == lane_id_asm());
    return id;
}

__device__ uint32_t lanemask_lt() { // TODO __lanemask_lt() not found
    return ((uint32_t)1 << lane_id()) - 1;
}

__device__ int active_lane_id(uint32_t active_mask) {
    return __popc(active_mask & lanemask_lt());
}

__device__ bool is_active(int lid, uint32_t active_mask) {
    return lid >= 0 && lid < 32 && (((uint32_t) 1) << lid) & active_mask;
}

/**
 * Align x to given alignment by padding if necessary
 */
__device__ size_t pad_align(size_t x, size_t alignment) {
    size_t mod = x % alignment;
    if (mod > 0) {
       return x + alignment - mod; 
    } else {
        return x;
    }
}

/**
 * assuming header can start at given offset from max-aligned starting point, compute where header will end
 *
 * @param offset Offset from max-aligned starting point
 * @param min_header_size Minimum header size required
 * @param payload_alignment Required alignment for payload.
 *
 * Requires payload_alignment == min_header_size || min_header_size == 8 && payload_alignment % min_header_size == 0
 * and sizeof(size_t) == 8 
 *
 * @return offset to end of header (and start of payload)
 * 
 */
__device__ size_t header_end_offset(size_t offset, size_t min_header_size, size_t payload_alignment) {
    assert(payload_alignment == min_header_size || min_header_size == 8 && payload_alignment % min_header_size == 0);
    assert(sizeof(size_t) == 8);
    size_t offset_initial = offset;

    // ensure that header start aligned by 2*x if header size = x < 8 (necessary for free)
    // -> header end (= payload start) not aligned by >x as shifted by x
    offset = pad_align(offset, min((size_t) 8, 2 * min_header_size));
    // offset now at position where header of block can start

    // write size of header
    offset += min_header_size;

    // due to precondition this further padding will only happen in the case min_header_size = 8
    // in which case it will not interfere with "misalignment requirements" which only exist for smaller headers
    offset = pad_align(offset, payload_alignment);

    assert(offset - offset_initial <= MAX_HEADER_PAD);
    return offset;
}

/**
 * compute minimum required header size and required alignemnt for a block
 *
 * @param size Requested payload size for the block
 * @param space_prev_block Space required for payload of previous block (including padding)
 *          has to be < 2^(64 - 3)
 *
 * @param res_min_header_size Will contain min_header_size
 * @param res_alignment Will contain alignment
 */
__device__ void compute_min_header_size_alignment(size_t size, size_t space_prev_block_payload, size_t& res_min_header_size, size_t& res_alignment) {
    // header has to be as big as alignment in order to conclude header size (or max of 8 bytes)
    // alignment has to as big as header in order to have legal position for header
    assert(sizeof(max_align_t) == 32); // TODO not even cuda's malloc aligns to 32

    // upper bound on space required by previous block where we count the padding of this blocks header too
    size_t bound_space_prev_block = space_prev_block_payload + MAX_HEADER_PAD;

    /* if (size < 2 && bound_space_prev_block < 32) { // 2 ^ (8-3) // TODO cannot use atomicCAS on char and on short only with compute capability >= 7
        // can 1-align header as no 2-alignment required
        res_alignment = 1;
        // can fit in 1 byte / 8 bits together with header
        res_min_header_size = 1;
    } else if (size < 4 &&  bound_space_prev_block < 8192) { // 2 ^ (16 - 3)
        res_alignment = 2;
        res_min_header_size = 2;
    } else */ if (size < 8 &&  bound_space_prev_block < 536870912) { // 2 ^ (32 - 3)
        res_alignment = 4;
        res_min_header_size = 4;
    } else if (size < 16) { // know that space_prev_block fits from initial check
        res_alignment = 8;
        res_min_header_size = 8;
    } else { // if (size < 32) { // TODO align to 32? not even cuda's malloc aligns to 32 but max_align_t is 32
        res_alignment = 16;
        res_min_header_size = 8; // header never bigger than 8
    } /*else {
        res_alignment = 32;
        res_min_header_size = 8;
    }*/
}

/**
 * write the header for a block.
 */
template<typename T>
__device__ void write_header(void* payload_start_ptr, bool is_superblock, bool is_lastblock, void* prev_payload_start_ptr) {
   T* header_ptr = ((T*) payload_start_ptr) - 1;
   size_t space_prev_payload = ((char*) header_ptr) - ((char*) prev_payload_start_ptr);  // includes padding

   T header = (T) space_prev_payload;

   assert(header == space_prev_payload);

   // write superblock bit
   header = header | (((T) is_superblock) << (8 * sizeof(T) - 1));

   // write lastblock bit
   header = header | (((T) is_lastblock) << (8 * sizeof(T) - 3));

   *header_ptr = header;
}


/**
 *  Warp level malloc with variable alignemnt and variable header size

 *  Each block has a header 
 *  where the first three  bits are is_superblock, is_free, and is_last_block
 *  and the remaining bits denote the size of the previous block
 *
 * 
 *  block 8-aligned <=> Header 8 byte
 *  block 4-aligned <=> Header 4 byte
 *  block 2-aligned <=> Header 2 byte
 *  block 1-aligned <=> Header 1 byte
 *
 */
__device__ void* malloc_v5(size_t size, bool coalesced) {
    
    assert(sizeof(size_t) == 8);
    // check that size < max size which is 2 ^ (64 - 3) - 1 as need to fit size in header together with extra bits
    if (size & (((size_t) 7) << 61) || size < 1) {
        return NULL;
    }

    int my_lane_id = lane_id();

    // retrieve mask of all threads in this warp that are currently executing
    // this instruction. they will perform a coalesced malloc
    uint32_t active_mask = __activemask();
    // count number of 1s
    int n_threads = __popc(active_mask);
    // Find the lowest-numbered active lane
    int elected_lane = __ffs(active_mask) - 1;
    bool is_elected = my_lane_id == elected_lane;
    // Find the highest-numbered active lane
    int last_lane = 31 - __clz(active_mask);
    bool is_last = my_lane_id == last_lane;
    // get id/idx among active lanes
    int my_active_lane_id = active_lane_id(active_mask);

    // compute relevant offsets from 16-bit aligned malloced superblock start
    size_t offset_prev_payload_end = 0; // offset to end of payload of last processed block;
    size_t offset_prev_payload_start = 0; // offset to end of header of last processed block (or start of payload, equivalent)
    for (int i = WARP_SIZE - 1; i > 0; i--) {  // go through all lanes/their memory  blocks from lowest lane to highest
        size_t size_i_below = __shfl_up_sync(active_mask, size, i);
        // check if result valid. if not both threads are active and participating
        // in shuffle, then result is undefined
        if (is_active(my_lane_id - i, active_mask)) {
            size_t min_header_size; size_t alignment;
            compute_min_header_size_alignment(size_i_below, offset_prev_payload_end - offset_prev_payload_start, min_header_size, alignment);
            offset_prev_payload_start = header_end_offset(offset_prev_payload_end, min_header_size, alignment);
            offset_prev_payload_end = offset_prev_payload_start + size;
        }
    }
    // arrived at own block, offset vars contain block of lane before

    // compute minimum header size and payload alignment for this block
    size_t min_header_size; size_t alignment;
    compute_min_header_size_alignment(size, offset_prev_payload_end - offset_prev_payload_start, min_header_size, alignment);
    size_t offset_payload_start = header_end_offset(offset_prev_payload_end, min_header_size, alignment);

    // let last thread compute required total length (= offset to end of its payload)
    size_t total_superblock_length = 0;
    if (is_last) {
        total_superblock_length = offset_payload_start + size;
    }
    total_superblock_length = __shfl_sync(active_mask, total_superblock_length, last_lane);


    // perform malloc of superblock
    char* malloced_ptr = NULL;
    // perform coalesced malloc
    if (is_elected) {
        malloced_ptr = (char*) malloc(total_superblock_length);
    }
    // broadcast alloced ptr to all lanes
    assert(sizeof(size_t) == sizeof(char*)); // make sure we don't change due to cast
    // need to cast as pointers can't be shuffled
    malloced_ptr = (char*) __shfl_sync(active_mask, (size_t) malloced_ptr, elected_lane);

    void* payload_start_ptr = malloced_ptr + offset_payload_start;
    void* prev_payload_start_ptr = malloced_ptr + offset_prev_payload_start;

    size_t payload_start_num = (size_t) payload_start_ptr;
    // work with correct header type
    if (payload_start_num % 8 == 0) {
        write_header<size_t>(payload_start_ptr, is_elected, is_last, prev_payload_start_ptr);
    } else if (payload_start_num % 4 == 0) {
        write_header<uint32_t>(payload_start_ptr, is_elected, is_last, prev_payload_start_ptr);
    } else {
        write_header<uint16_t>(payload_start_ptr, is_elected, is_last, prev_payload_start_ptr);
    }

    __syncwarp(active_mask); // required s.t. not uninitialized headers are looked at during free
    return payload_start_ptr;
}


typedef unsigned int min_h_t; // uint32_t header are the smallest headers we can use in compute capability 6

template<typename T>
__device__ min_h_t* read_header_templ(char* payload_start_ptr, size_t& size_result) {
   T* header_ptr = ((T*) payload_start_ptr) - 1;

   // read size
   size_result = (size_t) (*header_ptr & ~(((T) 7) << (8 * sizeof(T) - 3)));

   return (min_h_t*) header_ptr;
}


/**
 * read the header of a block
 * @param payload_start_ptr Ptr to start of payload
 * @param size_result Will hold size of prev block stored in header
 *
 * @return Pointer to start of header
 */
__device__ min_h_t* read_header(char* payload_start_ptr, size_t& size_result) {
    size_t payload_start_num = (size_t) payload_start_ptr;
    // work with correct header type
    if (payload_start_num % 8 == 0) {
        return read_header_templ<size_t>(payload_start_ptr, size_result);
    } else {// if (payload_start_num % 4 == 0) {
        return read_header_templ<uint32_t>(payload_start_ptr, size_result);
    } /*else { currently not supported
        return read_header_templ<uint16_t>(payload_start_ptr, size_result);
    }*/
}

/*
 * If not last, do nothing. 
 * Otherwise, traverse allcoated blocks until 
 *	- find block that is not freed -> set to be last block
 *	- find superblock (that is free) -> call free
 */
__device__ void free_v5(void* memptr) {

    min_h_t superblock_bit_mask = ((min_h_t) 1) << sizeof(min_h_t) - 1;
    min_h_t free_bit_mask = ((min_h_t) 1) << sizeof(min_h_t) - 2;
    min_h_t lastblock_bit_mask = ((min_h_t) 1) << sizeof(min_h_t) - 3;

    char* payload_start_ptr = (char*) memptr;
    size_t size_prev_block;
    min_h_t* header_start = read_header(payload_start_ptr, size_prev_block); // points to start of header (header might be larger than 16 bits)

    // mark block as free
    *header_start = *header_start | free_bit_mask;

    if (!(*header_start & lastblock_bit_mask)) {
        // block is not last block -> done (only last block does work
        return;
    }

    // from here on, we know that we have the last block
    // --> go through all prev blocks as described above

    min_h_t header_bits = *header_start;
    do {
        do {
            // payload_start_ptr, header_start, and header_bits contain freed block's info
            if (header_bits & superblock_bit_mask) {
                // if we reach the superblock and it's free we're done
                free(header_start);
                return;
            }
            // look at block before
            payload_start_ptr  = (((char*) header_start) - size_prev_block);
            min_h_t* header_start = read_header(payload_start_ptr, size_prev_block);
            header_bits = *header_start;
        } while (header_bits & free_bit_mask);

        // reached a non-free block -> try to set it to last block if it has not been modified inbetween
        // note that modified = freed here as no other modifications possible
    } while (atomicCAS(header_start, header_bits, (header_bits | lastblock_bit_mask)) != header_bits);
    // if the above CAS fails, we know that the block header has been modified -> block freed, and we
    // will continue walking through the free blocks

    // once we exit this loop we succeeded in setting an earlier unfreed block to be the last block -> we're done
}

/**
 * Safe warp level malloc
 *
 * Precondition: sizeof(size_t) == sizeof(void*) (given on our GPU)
 *
 *  Each block has a header of size sizeof(void*)
 *  with bits for is_superblock, is_free, and is_last_block
 *
 *  TODO check if threadsafe if concurrent frees in different warps/threadblocks of blocks malloced together
 *
 * TODO check current assumption:
 *     - lane_id cannot change
 *     - threads cannot move to different warps
 */
__device__ void* malloc_v4(size_t size, bool coalesced) {
    
    assert(sizeof(max_align_t) == 32);
    int alignment = 16; // TODO normal malloc doesn't necessarily align to 32 byte. Why? and doe we need to align to 16 even?

    size_t header_size_no_pad = sizeof(size_t);
    // pad to align 
    size_t header_size = alignment;
    assert(header_size >= header_size_no_pad);

    const size_t free_bit_mask = ((size_t) 1) << (8 * header_size_no_pad - 1);
    const size_t superblock_bit_mask = ((size_t) 1) << (8 * header_size_no_pad - 2);
    const size_t lastblock_bit_mask = ((size_t) 1) << (8 * header_size_no_pad - 3);

    // adjust size for alignment purposes
    if (size % alignment != 0) {
        size += alignment - size % alignment;
    }

    // assert special bits not used
    if ((free_bit_mask | superblock_bit_mask | lastblock_bit_mask) & size) {
        return NULL;
    }

    int my_lane_id = lane_id();

    // retrieve mask of all threads in this warp that are currently executing
    // this instruction. they will perform a coalesced malloc
    uint32_t active_mask = __activemask();
    // count number of 1s
    int n_threads = __popc(active_mask);
    // Find the lowest-numbered active lane
    int elected_lane = __ffs(active_mask) - 1;
    // get id/idx among active lanes
    int my_active_lane_id = active_lane_id(active_mask);

    // find out how much memory each thread needs
    size_t required_size_above = size; // how much all participating threads with lane_id >= own need
    // after step i, required_size_above holds the required size of next i threads
    // (including non-active threads for which the shuffle instruction returns 0)
    for (int i = 1; i < WARP_SIZE; i++) {
        size_t size_i_above = __shfl_down_sync(active_mask, size, i);
        // check if result valid. if not both threads are active and participating
        // in shuffle, then result is undefined
        if (is_active(my_lane_id + i, active_mask)) {
            required_size_above += size_i_above;
        }
    }

    __syncwarp(active_mask);

    // the elected_lane holds the total sum of required sizes
    size_t required_size_total = __shfl_sync(active_mask, required_size_above, elected_lane);

    char* malloced_ptr = NULL;

    // perform coalesced malloc
    if (my_lane_id == elected_lane) {
        malloced_ptr = (char*) malloc(required_size_total + n_threads * header_size);
        assert((long)malloced_ptr % alignment == 0);
    }

    // broadcast alloced ptr to all lanes
    assert(sizeof(size_t) == sizeof(char*)); // make sure we don't change due to cast
    // need to cast as pointers can't be shuffled
    malloced_ptr = (char*) __shfl_sync(active_mask, (size_t) malloced_ptr, elected_lane);

    // header space required for the threads with lower ids
    size_t header_size_below = my_active_lane_id * header_size;
    // compute this thread's memory region
    size_t* header_ptr = (size_t*) (malloced_ptr + required_size_total - required_size_above + header_size_below);

    // write headers

    // get size of participating block before
    size_t size_before = 0;
    bool found_size_before = false;
    for (int i = 1; i < WARP_SIZE; i++) {
        size_t size_i_below = __shfl_up_sync(active_mask, size, i);
        // check if result valid. if not both threads are active and participating
        // in shuffle, then result is undefined
        if (!found_size_before && is_active(my_lane_id - i, active_mask)) {
            size_before = size_i_below;
            found_size_before = true;
        }
    }
    if (my_lane_id == elected_lane) {
        // write superblock header
        *header_ptr = superblock_bit_mask;
    }  else {
        // write non-superblock header
        assert(found_size_before);
        *header_ptr = size_before;
    }


    // indicate last block
    if (my_active_lane_id == n_threads - 1) {
        *header_ptr = *header_ptr | lastblock_bit_mask;
    }

    // make sure that no blocks are returned for which neighboring blocks are not setup
    // as this could lead to problems when the returned blocks are freed
    __syncwarp(active_mask);

    return (void*) (((char*) header_ptr) + header_size);
}

/*
 * If not last, do nothing. 
 * Otherwise, traverse allcoated blocks until 
 *	- find block that is not freed -> set to be last block
 *	- find superblock (that is free) -> call free
 */
__device__ void free_v4(void* memptr) {
    
    assert(sizeof(size_t) == sizeof(long long unsigned int)); // required for cast in CAS call
    assert(sizeof(max_align_t) == 32);
    int alignment = 16; // TODO see above

    size_t header_size_no_pad = sizeof(size_t);
    // pad to align 
    size_t header_size = alignment;
    assert(header_size >= header_size_no_pad);

    const size_t free_bit_mask = ((size_t) 1) << (8 * header_size_no_pad - 1);
    const size_t superblock_bit_mask = ((size_t) 1) << (8 * header_size_no_pad - 2);
    const size_t lastblock_bit_mask = ((size_t) 1) << (8 * header_size_no_pad - 3);
    const size_t size_mask = ~ (free_bit_mask | superblock_bit_mask | lastblock_bit_mask);

    size_t* header_ptr = (size_t*) (((char*) memptr) - header_size);

    // set block to free
    *header_ptr = *header_ptr | free_bit_mask;

    if (!(*header_ptr & lastblock_bit_mask)) {
        return; // if we're not the last block, we're done
    }

    // from here on, we know that we have the last block
    // --> go through all prev blocks as described above

    size_t header = *header_ptr;
    do {
        do {
            // header ptr points to a freed block's header
            if (header & superblock_bit_mask) {
                // if we reach the superblock and it's free we're done
                free(header_ptr);
                return;
            }
            size_t size_prev_block = size_mask & header;
            header_ptr = (size_t*) (((char*) header_ptr) - header_size - size_prev_block);
            header = *header_ptr;
        } while (header & free_bit_mask);

        // reached a non-free block -> try to set it to last block if it has not been modified inbetween
        // note that modified = freed here as no other modifications possible
    } while (atomicCAS((long long unsigned int*) header_ptr, (long long unsigned int) header, (long long unsigned int) (header | lastblock_bit_mask)) != header);
    // if the above CAS fails, we know that the block header has been modified -> block freed, and we
    // will continue walking through the free blocks

    // once we exit this loop we succeeded in setting an earlier unfreed block to be the last block -> we're done
}