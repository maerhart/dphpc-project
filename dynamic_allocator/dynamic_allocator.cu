#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include "hip/hip_cooperative_groups.h"

// baseline using std malloc/free
__device__ void* malloc_baseline(size_t size) {
    void* ptr = malloc(size);
    #ifndef NDEBUG
    if (!ptr) {
        printf("GPUMPI: malloc failed to allocate %llu bytes on device\n", (long long unsigned)size);
    }
    #endif
    return ptr;
}

__device__ void free_baseline(void *memptr) {
    free(memptr);
}

// v1: allocate same sizes for future blocks

struct s_header {
    int counter;
};


// memory layout of a superblock
// s_header, blocksize x [pointer to s_header, data]
__device__ void* malloc_v1(size_t size) {
	__shared__ void* superblock;

	if (threadIdx.x == 0) {
		// allocate new superblock
		superblock = malloc(sizeof(s_header) + blockDim.x * (sizeof(s_header*) + size));
		if (!superblock) return NULL;
		
		// initialize header	
		struct s_header* header;
		header = (s_header*)superblock;
		header->counter = blockDim.x;
	}
	__syncthreads();

	// ptr to individual memory offset
    s_header* ptr = (s_header*)((char*)superblock + sizeof(s_header) + threadIdx.x * (size + sizeof(s_header*)));
	// set pointer to superblock header
	*ptr = *(s_header*)superblock;
	// return the pointer to the data section
	return (void*)(ptr + 1);
}

__device__ void free_v1(void* memptr) {
	// decrease counter
	s_header* header = (s_header*)memptr - 1;    
	int count = atomicSub(&(header->counter), 1);
	
	// last thread frees superblock
	if (count == 0) free(header);
}
