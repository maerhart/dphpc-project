#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include "hip/hip_cooperative_groups.h"

// baseline using std malloc/free
__device__ void* malloc_baseline(size_t size) {
    void* ptr = malloc(size);
    #ifndef NDEBUG
    if (!ptr) {
        printf("GPUMPI: malloc failed to allocate %llu bytes on device\n", (long long unsigned)size);
    }
    #endif
    return ptr;
}

__device__ void free_baseline(void *memptr) {
    free(memptr);
}

// v1: allocate same sizes for future blocks

struct s_header {
    int counter;
};


// memory layout of a superblock
// s_header, blocksize x [pointer to s_header, data]
__device__ void* malloc_v1(size_t size) {
	__shared__ void* superblock;

	if (threadIdx.x == 0) {
		// allocate new superblock
		superblock = malloc(sizeof(s_header) + blockDim.x * (sizeof(s_header*) + size));
		if (!superblock) return NULL;
		
		// initialize header	
		struct s_header* header;
		header = (s_header*)superblock;
		header->counter = blockDim.x;
	}
	__syncthreads();

	// ptr to individual memory offset
    s_header* ptr = (s_header*)((char*)superblock + sizeof(s_header) + threadIdx.x * (size + sizeof(s_header*)));
	// set pointer to superblock header
	*ptr = *(s_header*)superblock;
	// return the pointer to the data section
	return (void*)(ptr + 1);
}

__device__ void free_v1(void* memptr) {
	// decrease counter
	s_header* header = (s_header*)memptr - 1;    
	int count = atomicSub(&(header->counter), 1);
	
	// last thread frees superblock
	if (count == 0) free(header);
}



// allocate one int per thread and set to threadId
__global__ void test(int *resulting_ids) {
	
	int id = (blockIdx.x*blockDim.x + threadIdx.x);
	int* val = (int*)malloc_v1(sizeof(int));
	*val = id;
	resulting_ids[id] = *val;
}

int main(int argc, char* argv[]) {
	// run some simple unit tests, only in debug mode!
	int blocks = 100;
	int threads_per_block = 32;
	int total_threads = blocks * threads_per_block;

	int resulting_ids[total_threads];
	int *d_resulting_ids;
	hipMalloc(&d_resulting_ids, total_threads*sizeof(int));	

	test<<<blocks, threads_per_block>>>(d_resulting_ids);
	hipDeviceSynchronize(); // to allow for printf in kernel code
	hipMemcpy(resulting_ids, d_resulting_ids, total_threads*sizeof(int), hipMemcpyDeviceToHost);

	// sum up all ids, should match to the sum from 0 to total_threads
	bool passed = true;
	int sum = 0;
	for (int i = 0; i < total_threads; ++i) {
		sum += resulting_ids[i];
	}
	// sum up 0 to total_threads
	passed = sum == (total_threads - 1) * (total_threads) / 2;

	if (passed) {
		std::cout << "Tests passed" << std::endl;
	} 
	else {
		std::cout << "Failed" << std::endl;
	}

	return 0;
}
